
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

// CUDA-C includes
#include <hip/hip_runtime.h>


#include <cstdio>

#define DEBUG_CUDA 0
#define TESTPERFORMANCE_NO_LEVENSHTEIN 0

#define MIN3(a, b, c) \
    ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

int *d_numbersOfMatch;

__global__ void
searchPattern(char *buf, int n_bytes, char **pattern, int nb_patterns, int lastPatternAnalyzedByGPU, int *sizePatterns,
              int *numbersOfMatch, int indexFinishMyPieceWithoutExtra, int myRank, int numberProcesses,
              int indexStartMyPiece, int approx_factor) {

    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;

    // I analyze the second half of the patterns
    if (i < lastPatternAnalyzedByGPU) {

        if (TESTPERFORMANCE_NO_LEVENSHTEIN) {
            /*

             I should sleep for 1 microsecond

            The following code works just with Compute Capability >= 7.0
            unsigned int ns = 1000;
            __nanosleep(ns);

            Without the possibility to use nanosleep the only thing that it's possible to do is to wait an arbitrary number of clocks. But we don't know how many clocks correspond to a sleep of 1 microsecond.
            I could try through measurements to understand how many clocks correspond to 1 microsecond, but this is not so reliable. Different GPU can have different velocity (maybe one is running higher clock speed).

            clock_t start_clock = clock();
            clock_t clock_offset = 0;
            while (clock_offset < clock_count)
            {
                clock_offset = clock() - start_clock;
            }
            d_o[0] = clock_offset;

            */

        } else {

#if DEBUG_CUDA
            printf(
                            "MPI %d (out of %d). GPU: Started "
                            "to analize pattern n° %d.\n",
                            myRank, numberProcesses,
                            i);
#endif

            int sizeActualPattern = sizePatterns[i];

            int *column;
            column = (int *) malloc((sizeActualPattern + 1) * sizeof(int));
            if (column == NULL) {
                /*fprintf(
                        stderr,
                        "Error: unable to allocate memory for column (%ldB)\n",
                        (size_pattern + 1) * sizeof(int));
                // return 1;*/
            }

            // If I am not the last rank I should take in consideration
            // extra characters from the next piece: in this way I don't
            // miss words which are placed between two pieces. If am the
            // last rank I don't take extra characters as the other ranks
            // since the file is finished.
            int indexFinishMyPieceWithExtra =
                    indexFinishMyPieceWithoutExtra;
            if (myRank != numberProcesses - 1) {
                indexFinishMyPieceWithExtra += sizeActualPattern - 1;
            }

            // Traverse the input data up to the end of the file
            n_bytes = indexFinishMyPieceWithExtra;

            int r;
            for (r = indexStartMyPiece; r < n_bytes - approx_factor; r++) {

                int distance = 0;
                int size;
                size = sizeActualPattern;
                if (n_bytes - r < sizeActualPattern) {
                    size = n_bytes - r;
                }

                // I cannot call directly levenshtein function in GPU Code

                unsigned int x, y, lastdiag, olddiag;
                char * s1 = pattern[i];
                char *s2 = &buf[r];

#pragma unroll
                for (y = 1; y <= size; y++) {
                    column[y] = y;
                }
#pragma unroll
                for (x = 1; x <= size; x++) {
                    column[0] = x;
                    lastdiag = x - 1;
                    for (y = 1; y <= size; y++) {
                        olddiag = column[y];
                        column[y] = MIN3(column[y] + 1, column[y - 1] + 1,
                                         lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1));
                        lastdiag = olddiag;
                    }
                }

                distance = column[size];

                if (distance <= approx_factor) {
                    numbersOfMatch[i] += 1;

                }
            }

            free(column);
        }

    }

}


extern "C" int initializeGPU(char *buf, int n_bytes, char **pattern, int nb_patterns, int lastPatternAnalyzedByGPU,
                             int *sizePatterns, int indexFinishMyPieceWithoutExtra, int myRank, int numberProcesses,
                             int indexStartMyPiece, int approx_factor, int * numberOfMatchesInitialized) {

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Starting allocating data structures and memory transfers...\n");
#endif

    // I need to know the size of patterns to copy the data.
    // So I copy an array containing all the sizes of the patterns.
    int *d_sizePatterns;
    hipMalloc(&d_sizePatterns, nb_patterns * sizeof(int));
    hipMemcpy(d_sizePatterns, sizePatterns, nb_patterns * sizeof(int), hipMemcpyHostToDevice);

    // Allocate space for the buffer and copy data.
    char *d_buf;
    hipMalloc(&d_buf, n_bytes * sizeof(char));
    hipMemcpy(d_buf, buf, n_bytes * sizeof(char), hipMemcpyHostToDevice);

    // Allocate array where to save the number of matches
    hipHostMalloc(&d_numbersOfMatch, nb_patterns * sizeof(int), hipHostMallocDefault);
    hipMemcpy(d_numbersOfMatch, numberOfMatchesInitialized, nb_patterns * sizeof(int), hipMemcpyHostToDevice);

    // Allocate array of patterns: that is an array of arrays.
    // Need to use cudaMallocHost otherwise the following malloc throws a Segmentation Fault
    char **d_pattern;
    hipHostMalloc(&d_pattern, nb_patterns * sizeof(char *), hipHostMallocDefault);

    // Allocate space for each pattern and copy it
    for (int i = 0; i < nb_patterns; i++) {
        hipHostMalloc(&(d_pattern[i]), sizePatterns[i] * sizeof(char), hipHostMallocDefault);
        hipMemcpy(d_pattern[i], pattern[i], sizePatterns[i] * sizeof(char), hipMemcpyHostToDevice);
    }

    int sizeGrid = 256;
    int sizeBlocks = 10;

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Going to call the kernel code\n");
#endif

    searchPattern<<<sizeGrid, sizeBlocks>>>(d_buf, n_bytes, d_pattern, nb_patterns, lastPatternAnalyzedByGPU,
                                            d_sizePatterns, d_numbersOfMatch, indexFinishMyPieceWithoutExtra, myRank,
                                            numberProcesses, indexStartMyPiece, approx_factor);

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Kernel code returned.\n");
#endif

#if DEBUG_CUDA
    printf("CUDA_DEBUG. Copied results of CUDA.\n");
#endif

    return 1;

}

extern "C" int *
getGPUResult(int nb_patterns) {

    // Allocate local structure where to save the number of matches
    int *numbersOfMatch = (int *) malloc(nb_patterns * sizeof(int));

    // Copy the results from the GPU
    hipMemcpy(numbersOfMatch, d_numbersOfMatch, nb_patterns * sizeof(int),
               hipMemcpyDeviceToHost);

    return numbersOfMatch;
}